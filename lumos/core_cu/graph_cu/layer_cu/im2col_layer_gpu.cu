#include "im2col_layer_gpu.h"

void forward_im2col_layer_gpu(Layer l, int num)
{
    hipMemcpy(l.output, l.input, num*l.outputs*sizeof(float), hipMemcpyDeviceToDevice);
    float output_cpu[l.outputs];
    hipMemcpy(output_cpu, l.output, l.outputs*sizeof(float), hipMemcpyDeviceToHost);
}

void backward_im2col_layer_gpu(Layer l, float rate, int num, float *n_delta)
{
    hipMemcpy(l.delta, n_delta, num*l.inputs*sizeof(float), hipMemcpyDeviceToDevice);
}
